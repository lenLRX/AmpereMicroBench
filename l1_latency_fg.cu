#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <stdint.h>

// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define gpuErrchk(ans)                                                         \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

#define THREADS_NUM 1
#define WARP_SIZE 32

uint32_t *MakeChaseBuffer(uint32_t size) {
  uint32_t element_count = size / sizeof(uint32_t);
  uint32_t *result = new uint32_t[element_count + WARP_SIZE * 2];
  int round = element_count / WARP_SIZE;
  for (uint32_t i = 0; i < round; ++i) {
    for (uint32_t j = 0; j < WARP_SIZE; ++j) {
      result[i * WARP_SIZE + j] = 1;
    }
  }
  return result;
}

__global__ void l1_chase(uint32_t *duration, uint32_t *dsink,
                         uint32_t *posArray, uint32_t *l1_size_input) {
  uint32_t l1_size = l1_size_input[0];
  uint32_t iter_num = l1_size;
  // thread index
  uint32_t tid = threadIdx.x;
  // a register to avoid compiler optimization
  uint32_t sink = 0;
  // populate l1 cache to warm up
  for (uint32_t i = tid; i < l1_size; ++i) {
    uint32_t *ptr = (uint32_t *)posArray + i;
    uint32_t idx;
    asm volatile("{\t\n"
                 "ld.global.ca.u32 %0, [%1];\n\t"
                 "}"
                 : "=r"(idx)
                 : "l"(ptr)
                 : "memory");
    sink += idx;
  }
  // synchronize all threads
  asm volatile("bar.sync 0;");

  __shared__ uint32_t s_tvalue[4096];
  __shared__ uint32_t s_index[4096];

  posArray = posArray + tid;

  for (uint32_t i = 0; i < iter_num; ++i) {
    // start timing
    uint32_t start = 0;
    asm volatile("mov.u32 %0, %%clock;" : "=r"(start)::"memory");
    uint32_t idx;
    asm volatile("{\t\n"
                 "ld.global.ca.u32 %0, [%1];\n\t"
                 "}"
                 : "=r"(idx)
                 : "l"(posArray)
                 : "memory");
    posArray += idx;
    s_index[i] = idx;
    // stop timing
    uint32_t stop = 0;
    asm volatile("mov.u32 %0, %%clock;" : "=r"(stop)::"memory");
    s_tvalue[i] = stop - start;
  }
  // synchronize all threads
  asm volatile("bar.sync 0;");

  // write time and data back to memory
  for (uint32_t i = 0; i < iter_num; ++i) {
    duration[i] = s_tvalue[i];
    dsink[i] = s_index[i] + sink;
  }
}

void Testl1Latency(uint32_t test_cache_size, std::ofstream &ofs) {
  uint32_t l1_element_count = test_cache_size / sizeof(uint32_t);

  uint32_t *duration_host = new uint32_t[l1_element_count];
  uint32_t *chaseBuffer_host = MakeChaseBuffer(test_cache_size);

  uint32_t *chaseBuffer_dev;

  hipMalloc(&chaseBuffer_dev,
             (l1_element_count + WARP_SIZE) * sizeof(uint32_t));
  uint32_t *duration_dev;
  uint32_t *dsink_dev;
  uint32_t *l1_size_dev;
  hipMalloc(&duration_dev, l1_element_count * sizeof(uint32_t));
  hipMalloc(&dsink_dev, l1_element_count * sizeof(uint32_t));
  hipMalloc(&l1_size_dev, sizeof(uint32_t));

  hipMemcpy(l1_size_dev, &l1_element_count, sizeof(uint32_t),
             hipMemcpyHostToDevice);
  hipMemcpy(chaseBuffer_dev, chaseBuffer_host,
             (l1_element_count + WARP_SIZE) * sizeof(uint32_t),
             hipMemcpyHostToDevice);

  l1_chase<<<1, THREADS_NUM>>>(duration_dev, dsink_dev,
                               chaseBuffer_dev, l1_size_dev);

  gpuErrchk(hipDeviceSynchronize());
  hipMemcpy(duration_host, duration_dev, l1_element_count * sizeof(uint32_t),
             hipMemcpyDeviceToHost);

  int offset = 0;

  for (int i = 0; i < l1_element_count; ++i) {
    duration_host[i] -= 35;// 35 is overhead
    ofs << offset << "," << duration_host[i] << "\n";
    std::cout << offset << "," << duration_host[i] << "\n";
    offset += sizeof(uint32_t);
  }

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  double clock_rate = prop.clockRate * 1000;
  // std::cout << "Device freq:" << prop.clockRate << "khz" << std::endl;



  hipFree(chaseBuffer_dev);
  hipFree(duration_dev);
  hipFree(dsink_dev);
  hipFree(l1_size_dev);

  delete[] duration_host;
  delete[] chaseBuffer_host;
}

int main() {
  std::ofstream ofs("l1_latency_fg.csv");
  ofs << "offset(byte),latency(cycle)\n";
  Testl1Latency(8192, ofs);

}